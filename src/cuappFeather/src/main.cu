#include "hip/hip_runtime.h"
#include "main.cuh"

__global__ void hello_kernel() {
	printf("Hello from CUDA kernel!\\n");
}

void TestCUDA()
{
	hello_kernel << <1, 1 >> > ();
	hipDeviceSynchronize();
	std::cout << "CUDA finished\\n";
}

//#include <Serialization.hpp>

#include <nvtx3/nvToolsExt.h>

//int cuMain(const std::vector<float3>& host_points);
#include <hip/hip_runtime.h>
#include <>
#include <nvtx3/nvToolsExt.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <iostream>
#include <unordered_map>
#include <unordered_set>
#include <vector>
#include <algorithm>

namespace Clustering
{
	struct Voxel
	{
		float3 position;
		unsigned int label;
	};

	__global__ void Kernel_ClearVoxels(
		Voxel* d_voxels,
		unsigned int numberOfVoxels,
		dim3 volumeDimensions,
		float voxelSize,
		float3 volumeMin,
		float3 volumeCenter)
	{
		unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
		if (threadid >= volumeDimensions.x * volumeDimensions.y * volumeDimensions.z) return;

		d_voxels[threadid].position = make_float3(FLT_MAX, FLT_MAX, FLT_MAX);
		d_voxels[threadid].label = threadid;
	}

	void ClearVoxels(
		Voxel* d_voxels,
		unsigned int numberOfVoxels,
		dim3 volumeDimensions,
		float voxelSize,
		float3 volumeMin,
		float3 volumeCenter)
	{
		nvtxRangePushA("ClearVoxels");

		unsigned int blockSize = 256;
		unsigned int gridSize = (numberOfVoxels + blockSize - 1) / blockSize;
		Kernel_ClearVoxels << <gridSize, blockSize >> > (d_voxels, numberOfVoxels, volumeDimensions, voxelSize, volumeMin, volumeCenter);

		hipDeviceSynchronize();
		nvtxRangePop();
	}

	__global__ void Kernel_OccupyVoxels(
		float* d_points,
		unsigned int numberOfPoints,
		Voxel* d_voxels,
		unsigned int numberOfVoxels,
		dim3 volumeDimensions,
		float voxelSize,
		float3 volumeMin,
		float3 volumeCenter,
		uint3* occupiedVoxelIndices,
		unsigned int* numberOfOccupiedVoxelIndices,
		unsigned int* occupiedPointIndices,
		unsigned int* numberOfOccupiedPointIndices)
	{
		unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
		if (threadid >= numberOfPoints) return;

		auto gx = d_points[threadid * 3];
		auto gy = d_points[threadid * 3 + 1];
		auto gz = d_points[threadid * 3 + 2];

		if (gx < volumeMin.x || gx > volumeMin.x + volumeDimensions.x * voxelSize ||
			gy < volumeMin.y || gy > volumeMin.y + volumeDimensions.y * voxelSize ||
			gz < volumeMin.z || gz > volumeMin.z + volumeDimensions.z * voxelSize)
		{
			return;
		}

		unsigned int ix = (unsigned int)floorf((gx - volumeMin.x) / voxelSize);
		unsigned int iy = (unsigned int)floorf((gy - volumeMin.y) / voxelSize);
		unsigned int iz = (unsigned int)floorf((gz - volumeMin.z) / voxelSize);

		if (ix >= volumeDimensions.x || iy >= volumeDimensions.y || iz >= volumeDimensions.z) return;

		unsigned int volumeIndex = iz * volumeDimensions.x * volumeDimensions.y + iy * volumeDimensions.x + ix;
		auto& voxel = d_voxels[volumeIndex];

		voxel.position.x = volumeMin.x + ix * voxelSize;
		voxel.position.y = volumeMin.y + iy * voxelSize;
		voxel.position.z = volumeMin.z + iz * voxelSize;
		voxel.label = volumeIndex;

		//alog("%f, %f, %f\n", voxel.position.x, voxel.position.y, voxel.position.z);

		auto voxelIndex = atomicAdd(numberOfOccupiedVoxelIndices, 1);
		occupiedVoxelIndices[voxelIndex] = make_uint3(ix, iy, iz);

		auto pointIndex = atomicAdd(numberOfOccupiedPointIndices, 1);
		occupiedPointIndices[pointIndex] = threadid;

		//alog("%d\n", index);
	}

	void OccupyVoxels(
		float* d_points,
		unsigned int numberOfPoints,
		Voxel* d_voxels,
		unsigned int numberOfVoxels,
		dim3 volumeDimensions,
		float voxelSize,
		float3 volumeMin,
		float3 volumeCenter,
		uint3* occupiedVoxelIndices,
		unsigned int* numberOfOccupiedVoxelIndices,
		unsigned int* occupiedPointIndices,
		unsigned int* numberOfOccupiedPointIndices)
	{
		nvtxRangePush("OccupyVoxels");

		unsigned int blockSize = 256;
		unsigned int gridSize = (numberOfPoints + blockSize - 1) / blockSize;

		Kernel_OccupyVoxels << <gridSize, blockSize >> > (
			d_points,
			numberOfPoints,
			d_voxels,
			numberOfVoxels,
			volumeDimensions,
			voxelSize,
			volumeMin,
			volumeCenter,
			occupiedVoxelIndices,
			numberOfOccupiedVoxelIndices,
			occupiedPointIndices,
			numberOfOccupiedPointIndices);

		hipDeviceSynchronize();
		nvtxRangePop();
	}

	__device__ __forceinline__ unsigned int FindRoot(Voxel* d_voxels, unsigned int index)
	{
		while (d_voxels[index].label != index)
		{
			unsigned int parent = d_voxels[index].label;
			unsigned int grandparent = d_voxels[parent].label;

			if (parent != grandparent)
			{
				atomicCAS(&d_voxels[index].label, parent, grandparent);
			}
			index = d_voxels[index].label;
		}
		return index;
	}

	__device__ __forceinline__ void Union(Voxel* d_voxels, unsigned int a, unsigned int b)
	{
		unsigned int rootA = FindRoot(d_voxels, a);
		unsigned int rootB = FindRoot(d_voxels, b);

		if (rootA != rootB)
		{
			if (rootA < rootB)
				atomicMin(&d_voxels[rootB].label, rootA);
			else
				atomicMin(&d_voxels[rootA].label, rootB);
		}
	}

	__global__ void Kernel_ConnectedComponentLabeling(
		Voxel* d_voxels,
		uint3* occupiedVoxelIndices,
		unsigned int numberOfOccupiedVoxels,
		dim3 volumeDimensions)
	{
		unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
		if (threadid >= numberOfOccupiedVoxels) return;

		dim3 voxelIndex = occupiedVoxelIndices[threadid];
		unsigned int index = voxelIndex.z * volumeDimensions.x * volumeDimensions.y + voxelIndex.y * volumeDimensions.x + voxelIndex.x;

		if (d_voxels[index].position.x == FLT_MAX ||
			d_voxels[index].position.y == FLT_MAX ||
			d_voxels[index].position.z == FLT_MAX) return;

		int offset = 1;
		int xIndex = (int)voxelIndex.x;
		int yIndex = (int)voxelIndex.y;
		int zIndex = (int)voxelIndex.z;

		for (int zOffset = -offset; zOffset <= offset; zOffset++)
		{
			int nz = zIndex + zOffset;

			if (0 > nz || (int)volumeDimensions.z <= nz) continue;
			for (int yOffset = -offset; yOffset <= offset; yOffset++)
			{
				int ny = yIndex + yOffset;

				if (0 > ny || (int)volumeDimensions.y <= ny) continue;
				for (int xOffset = -offset; xOffset <= offset; xOffset++)
				{
					int nx = xIndex + xOffset;

					if (0 > nx || (int)volumeDimensions.x <= nx) continue;
					if (0 == xOffset && 0 == yOffset && 0 == zOffset) continue;


					if (nx >= 0 && nx < volumeDimensions.x &&
						ny >= 0 && ny < volumeDimensions.y &&
						nz >= 0 && nz < volumeDimensions.z)
					{
						unsigned int neighborIndex = nz * volumeDimensions.x * volumeDimensions.y + ny * volumeDimensions.x + nx;

						// Check if the neighbor is occupied
						if (d_voxels[neighborIndex].position.x != FLT_MAX)
						{
							Union(d_voxels, index, neighborIndex);
						}
					}
				}
			}
		}
	}

	void ConnectedComponentLabeling(
		Voxel* d_voxels,
		uint3* occupiedVoxelIndices,
		unsigned int numberOfOccupiedVoxelIndices,
		dim3 volumeDimensions)
	{
		nvtxRangePush("ConnectedComponentLabeling");

		unsigned int blockSize = 256;
		unsigned int gridSize = (numberOfOccupiedVoxelIndices + blockSize - 1) / blockSize;

		//for (int i = 0; i < 20; i++) // Increase iterations to ensure full convergence
		for (int i = 0; i < 2; i++)
		{
			Kernel_ConnectedComponentLabeling << <gridSize, blockSize >> > (
				d_voxels, occupiedVoxelIndices, numberOfOccupiedVoxelIndices, volumeDimensions);
			hipDeviceSynchronize();
		}

		hipDeviceSynchronize();
		nvtxRangePop();
	}

	__global__ void Kernel_GetLabels(
		float* d_points,
		unsigned int numberOfPoints,
		Voxel* d_voxels,
		unsigned int numberOfVoxels,
		dim3 volumeDimensions,
		float voxelSize,
		float3 volumeMin,
		float3 volumeCenter,
		unsigned int* d_labels)
	{
		unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
		if (threadid >= numberOfPoints) return;

		auto gx = d_points[threadid * 3];
		auto gy = d_points[threadid * 3 + 1];
		auto gz = d_points[threadid * 3 + 2];

		if (gx < volumeMin.x || gx > volumeMin.x + volumeDimensions.x * voxelSize ||
			gy < volumeMin.y || gy > volumeMin.y + volumeDimensions.y * voxelSize ||
			gz < volumeMin.z || gz > volumeMin.z + volumeDimensions.z * voxelSize)
		{
			return;
		}

		unsigned int ix = (unsigned int)floorf((gx - volumeMin.x) / voxelSize);
		unsigned int iy = (unsigned int)floorf((gy - volumeMin.y) / voxelSize);
		unsigned int iz = (unsigned int)floorf((gz - volumeMin.z) / voxelSize);

		if (ix >= volumeDimensions.x || iy >= volumeDimensions.y || iz >= volumeDimensions.z) return;

		unsigned int volumeIndex = iz * volumeDimensions.x * volumeDimensions.y + iy * volumeDimensions.x + ix;
		auto& voxel = d_voxels[volumeIndex];

		d_labels[threadid] = voxel.label;
	}

	std::vector<unsigned int> GetLabels(
		float* d_points,
		unsigned int numberOfPoints,
		Voxel* d_voxels,
		unsigned int numberOfVoxels,
		dim3 volumeDimensions,
		float voxelSize,
		float3 volumeMin,
		float3 volumeCenter)
	{
		unsigned int* d_labels = nullptr;
		hipMalloc(&d_labels, sizeof(unsigned int) * numberOfPoints);
		hipMemset(d_labels, -1, sizeof(unsigned int) * numberOfPoints);
		hipDeviceSynchronize();

		unsigned int blockSize = 256;
		unsigned int gridSize = (numberOfPoints + blockSize - 1) / blockSize;

		Kernel_GetLabels << <gridSize, blockSize >> > (
			d_points,
			numberOfPoints,
			d_voxels,
			numberOfVoxels,
			volumeDimensions,
			voxelSize,
			volumeMin,
			volumeCenter,
			d_labels);

		hipDeviceSynchronize();
		nvtxRangePop();

		std::vector<unsigned int> result(numberOfPoints);
		hipMemcpy(result.data(), d_labels, sizeof(unsigned int) * numberOfPoints, hipMemcpyDeviceToHost);
		hipDeviceSynchronize();

		hipFree(d_labels);

		return result;
	}

	void VisualizeVoxels(
		Voxel* d_voxels,
		unsigned int numberOfVoxels,
		dim3 volumeDimensions,
		float voxelSize,
		float3 volumeMin)
	{
		nvtxRangePush("VisualizeVoxels");

		Voxel* h_voxels = new Voxel[numberOfVoxels];
		hipMemcpy(h_voxels, d_voxels, sizeof(Voxel) * numberOfVoxels, hipMemcpyDeviceToHost);

		std::unordered_map<unsigned int, std::tuple<unsigned char, unsigned char, unsigned char>> labelToColor;

		std::unordered_map<unsigned int, unsigned int> labelHistogram;

		for (size_t i = 0; i < numberOfVoxels; i++)
		{
			auto& voxel = h_voxels[i];

			if (voxel.position.x != FLT_MAX) // Only visualize occupied voxels
			{
				unsigned int label = voxel.label;

				// Assign a unique color per label using a hash function
				if (labelToColor.find(label) == labelToColor.end())
				{
					unsigned char r = (label * 53) % 256;
					unsigned char g = (label * 97) % 256;
					unsigned char b = (label * 151) % 256;
					labelToColor[label] = std::make_tuple(r, g, b);
				}

				// Get the assigned color
				auto [r, g, b] = labelToColor[label];

				// Visualize the voxel with the computed color
 /*               VD::AddCube("labeled voxels", { voxel.position.x, voxel.position.y, voxel.position.z },
					0.05f, { r, g, b, 255 });*/

				if (0 == labelHistogram.count(voxel.label))
				{
					labelHistogram[voxel.label] = 1;
				}
				else
				{
					labelHistogram[voxel.label] += 1;
				}
			}
		}

		int i = 0;
		for (auto& [label, count] : labelHistogram)
		{
			alog("[%4d] voxel label - %16d : count - %8d\n", i++, label, count);
		}
		alog("\n");

		delete[] h_voxels;

		hipDeviceSynchronize();
		nvtxRangePop();
	}

	struct ClusteringCacheInfo
	{
		float voxelSize;
		dim3 cacheDimensions;
		unsigned int numberOfVoxels;
		float3 cacheMin;

		hipArray* cacheData3D = nullptr;
		hipSurfaceObject_t surfaceObject3D;

		uint3* occupiedVoxelIndices;
		unsigned int* numberOfOccupiedVoxelIndices;
	};
}


std::vector<unsigned int> cuMain(const std::vector<float3>& host_points, float3 center)
{
	nvtxRangePush("TestClustering");

	float* d_points = nullptr;
	hipMalloc(&d_points, sizeof(float) * host_points.size() * 3);
	hipMemcpy(d_points, host_points.data(), sizeof(float) * host_points.size() * 3, hipMemcpyHostToDevice);

	unsigned int numberOfPoints = host_points.size();
	dim3 volumeDimensions(400, 400, 400);
	unsigned int numberOfVoxels = volumeDimensions.x * volumeDimensions.y * volumeDimensions.z;
	float voxelSize = 0.1f;
	//float3 volumeCenter = make_float3(3.9904f, -15.8357f, -7.2774f);
	//float3 volumeCenter = make_float3(4.0f, -15.0f, -7.0f);
	float3 volumeCenter = make_float3(center.x, center.y, center.z);
	float3 volumeMin = make_float3(
		volumeCenter.x - (float)(volumeDimensions.x / 2) * voxelSize,
		volumeCenter.y - (float)(volumeDimensions.y / 2) * voxelSize,
		volumeCenter.z - (float)(volumeDimensions.z / 2) * voxelSize);

	Clustering::Voxel* d_voxels = nullptr;
	hipMalloc(&d_voxels, sizeof(Clustering::Voxel) * numberOfVoxels);

	uint3* occupiedVoxelIndices = nullptr;
	hipMalloc(&occupiedVoxelIndices, sizeof(uint3) * 5000000);
	unsigned int* numberOfOccupiedVoxelIndices = nullptr;
	hipMalloc(&numberOfOccupiedVoxelIndices, sizeof(unsigned int));
	hipMemset(numberOfOccupiedVoxelIndices, 0, sizeof(unsigned int));

	unsigned int* occupiedPointIndices = nullptr;
	hipMalloc(&occupiedPointIndices, sizeof(unsigned int) * 5000000);
	unsigned int* numberOfOccupiedPointIndices = nullptr;
	hipMalloc(&numberOfOccupiedPointIndices, sizeof(unsigned int));
	hipMemset(numberOfOccupiedPointIndices, 0, sizeof(unsigned int));

	hipDeviceSynchronize();

	ClearVoxels(d_voxels, numberOfVoxels, volumeDimensions, voxelSize, volumeMin, volumeCenter);

	OccupyVoxels(
		d_points,
		numberOfPoints,
		d_voxels,
		numberOfVoxels,
		volumeDimensions,
		voxelSize,
		volumeMin,
		volumeCenter,
		occupiedVoxelIndices,
		numberOfOccupiedVoxelIndices,
		occupiedPointIndices,
		numberOfOccupiedPointIndices);

	unsigned int h_numberOfOccupiedVoxelIndices = 0;
	hipMemcpy(&h_numberOfOccupiedVoxelIndices, numberOfOccupiedVoxelIndices, sizeof(unsigned int), hipMemcpyDeviceToHost);

	ConnectedComponentLabeling(d_voxels, occupiedVoxelIndices, h_numberOfOccupiedVoxelIndices, volumeDimensions);

	VisualizeVoxels(
		d_voxels,
		numberOfVoxels,
		volumeDimensions,
		voxelSize,
		volumeMin);

	std::vector<unsigned int> result = GetLabels(
		d_points,
		numberOfPoints,
		d_voxels,
		numberOfVoxels,
		volumeDimensions,
		voxelSize,
		volumeMin,
		volumeCenter);

	{
		std::unordered_map<unsigned int, unsigned int> labelHistogram;

		for (auto& i : result)
		{
			if (0 == labelHistogram.count(i))
			{
				labelHistogram[i] = 1;
			}
			else
			{
				labelHistogram[i] += 1;
			}
		}

		unsigned int i = 0;
		for (auto& [label, count] : labelHistogram)
		{
			alog("[%4d] point label - %16d : count - %8d\n", i++, label, count);
		}
		alog("\n");
	}

	unsigned int h_numberOfOccupiedPointIndices = 0;
	hipMemcpy(&h_numberOfOccupiedPointIndices, numberOfOccupiedPointIndices, sizeof(unsigned int), hipMemcpyDeviceToHost);
	unsigned int* h_occupiedPointIndices = new unsigned int[h_numberOfOccupiedPointIndices];
	hipMemcpy(h_occupiedPointIndices, occupiedPointIndices, sizeof(unsigned int) * h_numberOfOccupiedPointIndices, hipMemcpyDeviceToHost);

	for (size_t i = 0; i < h_numberOfOccupiedPointIndices; i++)
	{
		auto index = h_occupiedPointIndices[i];
		auto p = host_points[index];

		//VD::AddSphere("In Area", { x,y, z }, 0.05f, { 255, 0, 0 });
	}

	hipFree(d_points);
	hipFree(d_voxels);
	hipFree(occupiedVoxelIndices);
	hipFree(numberOfOccupiedVoxelIndices);
	hipFree(occupiedPointIndices);
	hipFree(numberOfOccupiedPointIndices);

	delete[] h_occupiedPointIndices;

	hipDeviceSynchronize();
	nvtxRangePop();

	return result;
}