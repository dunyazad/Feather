#include "hip/hip_runtime.h"
#include "main.cuh"

__global__ void hello_kernel() {
    printf("Hello from CUDA kernel!\\n");
}

void TestCUDA()
{
    hello_kernel << <1, 1 >> > ();
    hipDeviceSynchronize();
    std::cout << "CUDA finished\\n";
}
